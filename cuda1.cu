#include "hip/hip_runtime.h"
#include "parameters.h"
#include <stdio.h>
__global__ void cuda_kernel(float *B, float *A, IndexSave *dInd)
{
	// complete cuda kernel function
	int TotalThread = blockDim.x * gridDim.x;
	int stripe = SIZE / TotalThread;
	int head = (blockIdx.x * blockDim.x + threadIdx.x) * stripe;
	int LoopLim = head + stripe;

	for (int i = head; i < LoopLim; i++)
	{
		dInd[i].blockInd_x = blockIdx.x;
		dInd[i].threadInd_x = threadIdx.x;
		dInd[i].head = head;
		dInd[i].stripe = stripe;

		B[i] = (B[i] - A[i]) * (B[i] - A[i]);
	}
};

float GPU_kernel(float *B, float *A, IndexSave *indsave)
{
	float *dA, *dB;
	IndexSave *dInd;

	// Creat Timing Event
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate Memory Space on Device

	// Allocate Memory Space on Device (for observation)
	hipMalloc((void **)&dInd, sizeof(IndexSave) * SIZE);

	// Copy Data to be Calculated
	hipMalloc((void **)&dB, sizeof(float) * SIZE);
	hipMalloc((void **)&dA, sizeof(float) * SIZE);
	// Copy Data to be Calculated
	hipMemcpy(dB, B, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dA, A, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	// Copy Data (indsave array) to device
	hipMemcpy(dInd, indsave, sizeof(IndexSave) * SIZE, hipMemcpyHostToDevice);

	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid(4);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid, dimBlock>>>(dB, dA, dInd);
	// Stop Timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Copy Output back
	hipMemcpy(B, dB, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(A, dA, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(indsave, dInd, sizeof(IndexSave) * SIZE, hipMemcpyDeviceToHost);

	// Release Memory Space on Device
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	return elapsedTime;
}
