#include <stdio.h>

#include "parameters.h"

extern float GPU_kernel(float *B, float *A, IndexSave *indsave);

void genNumbers(float *A, float *B, int size)
{

	for (int i = 0; i < size; i++)
	{
		A[i] = 1.0 * ((rand() % 256) / 256.0);
		B[i] = 1.0 * ((rand() % 256) / 256.0);
	}
}

void function_1(float *B, float *A, float *C)
{

	for (int i = 0; i < SIZE; i++)
	{
		C[i] = (B[i] - A[i]) * (B[i] - A[i]);
	}
}

bool verify(float a, float b)
{
	if (a != b)
		return true;
	return false;
}

void printIndex(IndexSave *indsave, float *B, float *C, float *a, float *b)
{
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d,im here4!\n", i);
		printf("%d : blockInd_x=%d,threadInd_x=%d,head=%d,stripe=%d", i, (indsave[i]).blockInd_x, (indsave[i]).threadInd_x, (indsave[i]).head, (indsave[i]).stripe);
		printf(" || GPU result=%f,CPU result=%f\n", B[i], C[i]);
		*a += C[i];
		*b += B[i];
	}
}

int main()

{
	// random seed
	float *A = new float[SIZE];
	// random number sequence computed by GPU
	float *B = new float[SIZE];
	// random number sequence computed by CPU
	float *C = new float[SIZE];
	// Indices saver (for checking correctness)
	IndexSave *indsave = new IndexSave[SIZE];

	genNumbers(A, B, SIZE);

	/* CPU side*/
	function_1(B, A, C);

	/* GPU side*/
	float elapsedTime = GPU_kernel(B, A, indsave);
	float lossc = 0;
	float lossg = 0;

	/*Show threads execution info*/
	printIndex(indsave, B, C, &lossc, &lossg);

	printf("==============================================\n");
	/* verify the result*/
	if (verify(lossg, lossc))
	{
		printf("wrong answer\n");
	}
	printf("GPU time = %5.2f ms\n", elapsedTime);

	/*Please press any key to exit the program*/
	getchar();
}
